﻿#include <iostream>
#include <fstream>
#include <chrono>
#include <cmath>
#include <cfloat>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define NB_ELT 2
#define NB_TEST 1000000ll
#define NB_PAS_MAX 81
#define TPS_MAX_PAR_TEST 10
#define MIN_CLCK 1


#ifdef _WIN32
#include <windows.h>
#include <processthreadsapi.h>
FILETIME a, b, c, d;
inline double getCpuTime()
{
	if (GetProcessTimes(GetCurrentProcess(), &a, &b, &c, &d) != 0)
    {
		return
			(double)(d.dwLowDateTime |
				((unsigned long long)d.dwHighDateTime << 32)) * 0.0000001;
	}
	return 0;
}
#else
inline double getCpuTime()
{
	return std::clock() / double(CLOCKS_PER_SEC);
}
#endif


__global__ void initTab(double* tab, double val, int nbElt)
{
	for (int idx = 0; idx < nbElt; idx++)
		tab[idx] = val+idx;
}

__global__ void addTestLoop(double* tabA, double* tabB, double* tabC, int nbElt, int nbTest)
{
	for (long long int idxTest = 0; idxTest < nbTest; idxTest++)
		for (int i = 0; i < nbElt; i++)
			tabC[i] = tabA[i] + tabB[i];

}
 

int main() {
	int deviceId;
	int numberOfSMs;
	int deviceCount = 0;
	hipError_t erreur;
	hipGetDeviceCount(&deviceCount);

	hipGetDevice(&deviceId);
	hipSetDevice(deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
	size_t free, total;
	hipError_t cuRes=hipMemGetInfo(&free, &total);
	erreur = hipGetLastError();
	if (erreur != hipSuccess)
		std::cout << "Error: " << hipGetErrorString(erreur) << "\n";
	std::cout << "free memory : " << free << "\n";
	std::cout << "total memory : " << total << "\n";


	int nbPas = int(std::log(std::min(int(free / 48), int(pow(2.0, NB_PAS_MAX / 3.0)))) / std::log(2))*3;
	std::ofstream rapport("tps_fct_mem.txt");
	double tpsPre = 0;
	long long int nbTest = NB_TEST;
	int nbEltMax = int(pow(2.0, nbPas / 3.0));
	double *tabA, *tabB, *tabC;
	hipMallocManaged(&tabA, sizeof(double) * nbEltMax);
	erreur = hipGetLastError();
	if (erreur != hipSuccess)
		std::cout << "Error: " << hipGetErrorString(erreur) << "\n";
	hipMallocManaged(&tabB, sizeof(double) * nbEltMax);
	erreur = hipGetLastError();
	if (erreur != hipSuccess)
		std::cout << "Error: " << hipGetErrorString(erreur) << "\n";
	hipMallocManaged(&tabC, sizeof(double) * nbEltMax);
	erreur = hipGetLastError();
	if (erreur != hipSuccess)
		std::cout << "Error: " << hipGetErrorString(erreur) << "\n";
	for (int idx = 7; idx < nbPas;idx++)
	{
		
		int nbElt = NB_ELT * int(pow(2.0, idx / 3.0));
		initTab <<<1, 1 >>> (tabA, 2.0, nbElt);
		erreur = hipGetLastError();
		if (erreur != hipSuccess)
			std::cout << "Error: " << hipGetErrorString(erreur) << "\n";
		initTab <<<1, 1 >>> (tabB, 3.0, nbElt);
		erreur = hipGetLastError();
		if (erreur != hipSuccess)
			std::cout << "Error: " << hipGetErrorString(erreur) << "\n";
		initTab <<<1, 1 >>> (tabC, 0, nbElt);
		erreur = hipGetLastError();
		if (erreur != hipSuccess)
			std::cout << "Error: " << hipGetErrorString(erreur) << "\n";
		double tpsParTest = 0;
		if (tpsPre > TPS_MAX_PAR_TEST)
			nbTest /= 2;
		if (nbTest == 0)
			nbTest = 1;
		rapport << nbElt << "\t" << nbTest << "\t";
		double finPre;
		double debut = getCpuTime();
		double tpsMin = DBL_MAX;
		addTestLoop <<<1, 1 >>> (tabA, tabB, tabC, nbElt, nbTest);
		hipDeviceSynchronize();
		erreur = hipGetLastError();
		if (erreur != hipSuccess)
			std::cout << "Error: " << hipGetErrorString(erreur) << "\n";
		finPre = getCpuTime();
		double tps = finPre - debut;
		tpsParTest = tps;
		std::cout << "<-- " << nbElt << " -->\nDurée sans thread (" << tpsParTest << " ticks) ";
		tpsPre = tps;
		tpsParTest = tpsParTest  / nbTest;
		std::cout << tpsParTest << "s (" << tpsParTest / nbElt << "s par élément) nbTest=" << nbTest<<"\n";
		rapport << tpsParTest / nbElt << "\t" << tps << "\t" << tpsMin / nbElt ;
		rapport << "\n";
		rapport.flush();

	}
	delete tabA;
	delete tabB;
	delete tabC;
	rapport.close();
    return 0;
}

